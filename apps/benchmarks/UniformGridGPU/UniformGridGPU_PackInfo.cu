#include "hip/hip_runtime.h"
#include "stencil/Directions.h"
#include "core/cell/CellInterval.h"
#include "cuda/GPUField.h"
#include "core/DataTypes.h"
#include "UniformGridGPU_PackInfo.h"


#define FUNC_PREFIX __global__


namespace walberla {
namespace pystencils {

using walberla::cell::CellInterval;
using walberla::stencil::Direction;



namespace internal_pack_SW {
static FUNC_PREFIX void pack_SW(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_BW {
static FUNC_PREFIX void pack_BW(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_W {
static FUNC_PREFIX void pack_W(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 3*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_33[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_TW {
static FUNC_PREFIX void pack_TW(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_NW {
static FUNC_PREFIX void pack_NW(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_BS {
static FUNC_PREFIX void pack_BS(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_S {
static FUNC_PREFIX void pack_S(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_32 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 2*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_32[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_TS {
static FUNC_PREFIX void pack_TS(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_B {
static FUNC_PREFIX void pack_B(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_36 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 6*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_36[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_C {
static FUNC_PREFIX void pack_C(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_30[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_T {
static FUNC_PREFIX void pack_T(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_35 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 5*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_35[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_BN {
static FUNC_PREFIX void pack_BN(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_N {
static FUNC_PREFIX void pack_N(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_31 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + _stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_31[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_TN {
static FUNC_PREFIX void pack_TN(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_SE {
static FUNC_PREFIX void pack_SE(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_BE {
static FUNC_PREFIX void pack_BE(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_E {
static FUNC_PREFIX void pack_E(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x] = _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1] = _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2] = _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 4*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3] = _data_pdfs_10_20_34[_stride_pdfs_0*ctr_0];
      double * const _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4] = _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_TE {
static FUNC_PREFIX void pack_TE(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0];
   } 
}
}

namespace internal_pack_NE {
static FUNC_PREFIX void pack_NE(double * _data_buffer, double * const _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * const _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x] = _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0];
   } 
}
}



namespace internal_unpack_NE {
static FUNC_PREFIX void unpack_NE(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_TE {
static FUNC_PREFIX void unpack_TE(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_E {
static FUNC_PREFIX void unpack_E(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 3*_stride_pdfs_3;
      _data_pdfs_10_20_33[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_BE {
static FUNC_PREFIX void unpack_BE(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_SE {
static FUNC_PREFIX void unpack_SE(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_TN {
static FUNC_PREFIX void unpack_TN(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_N {
static FUNC_PREFIX void unpack_N(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_32 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 2*_stride_pdfs_3;
      _data_pdfs_10_20_32[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_39 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 9*_stride_pdfs_3;
      _data_pdfs_10_20_39[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_BN {
static FUNC_PREFIX void unpack_BN(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_T {
static FUNC_PREFIX void unpack_T(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_316 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 16*_stride_pdfs_3;
      _data_pdfs_10_20_316[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_317 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 17*_stride_pdfs_3;
      _data_pdfs_10_20_317[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_36 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 6*_stride_pdfs_3;
      _data_pdfs_10_20_36[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_C {
static FUNC_PREFIX void unpack_C(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2;
      _data_pdfs_10_20_30[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_B {
static FUNC_PREFIX void unpack_B(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_312 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 12*_stride_pdfs_3;
      _data_pdfs_10_20_312[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_313 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 13*_stride_pdfs_3;
      _data_pdfs_10_20_313[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_35 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 5*_stride_pdfs_3;
      _data_pdfs_10_20_35[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_TS {
static FUNC_PREFIX void unpack_TS(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_S {
static FUNC_PREFIX void unpack_S(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_31 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + _stride_pdfs_3;
      _data_pdfs_10_20_31[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_315 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 15*_stride_pdfs_3;
      _data_pdfs_10_20_315[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_37 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 7*_stride_pdfs_3;
      _data_pdfs_10_20_37[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_BS {
static FUNC_PREFIX void unpack_BS(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_311 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 11*_stride_pdfs_3;
      _data_pdfs_10_20_311[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_NW {
static FUNC_PREFIX void unpack_NW(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_TW {
static FUNC_PREFIX void unpack_TW(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_W {
static FUNC_PREFIX void unpack_W(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_310 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 10*_stride_pdfs_3;
      _data_pdfs_10_20_310[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x];
      double * _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 1];
      double * _data_pdfs_10_20_318 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 18*_stride_pdfs_3;
      _data_pdfs_10_20_318[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 2];
      double * _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 4*_stride_pdfs_3;
      _data_pdfs_10_20_34[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 3];
      double * _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(5*blockDim.z*blockIdx.z + 5*threadIdx.z) + _size_pdfs_0*(5*blockDim.y*blockIdx.y + 5*threadIdx.y) + 5*blockDim.x*blockIdx.x + 5*threadIdx.x + 4];
   } 
}
}

namespace internal_unpack_BW {
static FUNC_PREFIX void unpack_BW(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_314 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 14*_stride_pdfs_3;
      _data_pdfs_10_20_314[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}

namespace internal_unpack_SW {
static FUNC_PREFIX void unpack_SW(double * const _data_buffer, double * _data_pdfs, int64_t const _size_pdfs_0, int64_t const _size_pdfs_1, int64_t const _size_pdfs_2, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < _size_pdfs_0 && blockDim.y*blockIdx.y + threadIdx.y < _size_pdfs_1 && blockDim.z*blockIdx.z + threadIdx.z < _size_pdfs_2)
   {
      const int64_t ctr_0 = blockDim.x*blockIdx.x + threadIdx.x;
      const int64_t ctr_1 = blockDim.y*blockIdx.y + threadIdx.y;
      const int64_t ctr_2 = blockDim.z*blockIdx.z + threadIdx.z;
      double * _data_pdfs_10_20_38 = _data_pdfs + _stride_pdfs_1*ctr_1 + _stride_pdfs_2*ctr_2 + 8*_stride_pdfs_3;
      _data_pdfs_10_20_38[_stride_pdfs_0*ctr_0] = _data_buffer[_size_pdfs_0*_size_pdfs_1*(blockDim.z*blockIdx.z + threadIdx.z) + _size_pdfs_0*(blockDim.y*blockIdx.y + threadIdx.y) + blockDim.x*blockIdx.x + threadIdx.x];
   } 
}
}




void UniformGridGPU_PackInfo::pack(Direction dir, unsigned char * byte_buffer, IBlock * block, hipStream_t stream)
{
    double * buffer = reinterpret_cast<double*>(byte_buffer);

    auto pdfs = block->getData< cuda::GPUField<double> >(pdfsID);

    CellInterval ci;
    pdfs->getSliceBeforeGhostLayer(dir, ci, 1, false);

    switch( dir )
    {
        case stencil::SW:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_SW::pack_SW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BW:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_BW::pack_BW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::W:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_W::pack_W<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TW:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_TW::pack_TW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::NW:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_NW::pack_NW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BS:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_BS::pack_BS<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::S:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_S::pack_S<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TS:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_TS::pack_TS<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::B:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_B::pack_B<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::C:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_C::pack_C<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2);
            break;
        }
        
        case stencil::T:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_T::pack_T<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BN:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_BN::pack_BN<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::N:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_N::pack_N<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TN:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_TN::pack_TN<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::SE:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_SE::pack_SE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BE:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_BE::pack_BE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::E:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_E::pack_E<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TE:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_TE::pack_TE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::NE:
        {
            double * _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * const _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_pack_NE::pack_NE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        

        default:
            WALBERLA_ASSERT(false);
    }
}


void UniformGridGPU_PackInfo::unpack(Direction dir, unsigned char * byte_buffer, IBlock * block, hipStream_t stream)
{
    double * buffer = reinterpret_cast<double*>(byte_buffer);

    auto pdfs = block->getData< cuda::GPUField<double> >(pdfsID);

    CellInterval ci;
    pdfs->getGhostRegion(dir, ci, 1, false);

    switch( dir )
    {
        case stencil::NE:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_NE::unpack_NE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TE:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_TE::unpack_TE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::E:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_E::unpack_E<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BE:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_BE::unpack_BE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::SE:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_SE::unpack_SE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TN:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_TN::unpack_TN<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::N:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_N::unpack_N<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BN:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_BN::unpack_BN<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::T:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_T::unpack_T<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::C:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_C::unpack_C<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2);
            break;
        }
        
        case stencil::B:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_B::unpack_B<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TS:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_TS::unpack_TS<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::S:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_S::unpack_S<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BS:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_BS::unpack_BS<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::NW:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_NW::unpack_NW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::TW:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_TW::unpack_TW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::W:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_W::unpack_W<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::BW:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_BW::unpack_BW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        
        case stencil::SW:
        {
            double * const _data_buffer = buffer;
            WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(pdfs->nrOfGhostLayers()));
            WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(pdfs->nrOfGhostLayers()));
            double * _data_pdfs = pdfs->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->xSizeWithGhostLayer(), int64_t(cell_idx_c(ci.xSize()) + 0));
            const int64_t _size_pdfs_0 = int64_t(cell_idx_c(ci.xSize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->ySizeWithGhostLayer(), int64_t(cell_idx_c(ci.ySize()) + 0));
            const int64_t _size_pdfs_1 = int64_t(cell_idx_c(ci.ySize()) + 0);
            WALBERLA_ASSERT_GREATER_EQUAL(pdfs->zSizeWithGhostLayer(), int64_t(cell_idx_c(ci.zSize()) + 0));
            const int64_t _size_pdfs_2 = int64_t(cell_idx_c(ci.zSize()) + 0);
            const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
            const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
            const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
            const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
            dim3 _block(int(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)), int(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)), int(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)));
            dim3 _grid(int(( (_size_pdfs_0) % (((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) == 0 ? (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) : ( (int64_t)(_size_pdfs_0) / (int64_t)(((16 < _size_pdfs_0) ? 16 : _size_pdfs_0)) ) +1 )), int(( (_size_pdfs_1) % (((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) == 0 ? (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) : ( (int64_t)(_size_pdfs_1) / (int64_t)(((16 < _size_pdfs_1) ? 16 : _size_pdfs_1)) ) +1 )), int(( (_size_pdfs_2) % (((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) == 0 ? (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) : ( (int64_t)(_size_pdfs_2) / (int64_t)(((1 < _size_pdfs_2) ? 1 : _size_pdfs_2)) ) +1 )));
            internal_unpack_SW::unpack_SW<<<_grid, _block, 0, stream>>>(_data_buffer, _data_pdfs, _size_pdfs_0, _size_pdfs_1, _size_pdfs_2, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3);
            break;
        }
        

        default:
            WALBERLA_ASSERT(false);
    }
}


uint_t UniformGridGPU_PackInfo::size(stencil::Direction dir, IBlock * block)
{
    auto pdfs = block->getData< cuda::GPUField<double> >(pdfsID);

    CellInterval ci;
    pdfs->getGhostRegion(dir, ci, 1, false);

    uint_t elementsPerCell = 0;

    switch( dir )
    {
        case stencil::SW:
            elementsPerCell = 1;
            break;
        
        case stencil::BW:
            elementsPerCell = 1;
            break;
        
        case stencil::W:
            elementsPerCell = 5;
            break;
        
        case stencil::TW:
            elementsPerCell = 1;
            break;
        
        case stencil::NW:
            elementsPerCell = 1;
            break;
        
        case stencil::BS:
            elementsPerCell = 1;
            break;
        
        case stencil::S:
            elementsPerCell = 5;
            break;
        
        case stencil::TS:
            elementsPerCell = 1;
            break;
        
        case stencil::B:
            elementsPerCell = 5;
            break;
        
        case stencil::C:
            elementsPerCell = 1;
            break;
        
        case stencil::T:
            elementsPerCell = 5;
            break;
        
        case stencil::BN:
            elementsPerCell = 1;
            break;
        
        case stencil::N:
            elementsPerCell = 5;
            break;
        
        case stencil::TN:
            elementsPerCell = 1;
            break;
        
        case stencil::SE:
            elementsPerCell = 1;
            break;
        
        case stencil::BE:
            elementsPerCell = 1;
            break;
        
        case stencil::E:
            elementsPerCell = 5;
            break;
        
        case stencil::TE:
            elementsPerCell = 1;
            break;
        
        case stencil::NE:
            elementsPerCell = 1;
            break;
        
        default:
            elementsPerCell = 0;
    }
    return ci.numCells() * elementsPerCell * sizeof( double );
}



} // namespace pystencils
} // namespace walberla