#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file UniformGridGPU_NoSlip.cpp
//! \\ingroup lbm
//! \\author lbmpy
//======================================================================================================================

#include <cmath>

#include "core/DataTypes.h"
#include "core/Macros.h"
#include "UniformGridGPU_NoSlip.h"
#include "cuda/ErrorChecking.h"


#define FUNC_PREFIX __global__

using namespace std;

namespace walberla {
namespace lbm {

#ifdef __GNUC__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wunused-variable"
#pragma GCC diagnostic ignored "-Wconversion"
#endif

#ifdef __HIPCC__
#pragma push
#pragma diag_suppress = declared_but_not_referenced
#endif


namespace internal_boundary_UniformGridGPU_NoSlip {
static FUNC_PREFIX void boundary_UniformGridGPU_NoSlip(uint8_t * const _data_indexVector, double * _data_pdfs, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, int64_t indexVectorSize)
{
   if (blockDim.x*blockIdx.x + threadIdx.x < indexVectorSize)
   {
      uint8_t * const _data_indexVector_10 = _data_indexVector;
      const int32_t x = *((int32_t *)(& _data_indexVector_10[16*blockDim.x*blockIdx.x + 16*threadIdx.x]));
      uint8_t * const _data_indexVector_14 = _data_indexVector + 4;
      const int32_t y = *((int32_t *)(& _data_indexVector_14[16*blockDim.x*blockIdx.x + 16*threadIdx.x]));
      uint8_t * const _data_indexVector_18 = _data_indexVector + 8;
      const int32_t z = *((int32_t *)(& _data_indexVector_18[16*blockDim.x*blockIdx.x + 16*threadIdx.x]));
      
      
      const int64_t cx [] = { 0, 0, 0, -1, 1, 0, 0, -1, 1, -1, 1, 0, 0, -1, 1, 0, 0, -1, 1 };
      const int64_t cy [] = { 0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, -1, 0, 0, 1, -1, 0, 0 };
      const int64_t cz [] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, 1, 1, -1, -1, -1, -1 };
      const int invdir [] = { 0, 2, 1, 4, 3, 6, 5, 10, 9, 8, 7, 16, 15, 18, 17, 12, 11, 14, 13 };
      
      
      const double weights [] = { 0.333333333333333,0.0555555555555556,0.0555555555555556,0.0555555555555556,0.0555555555555556,0.0555555555555556,0.0555555555555556,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778,0.0277777777777778 };
      
      uint8_t * const _data_indexVector_112 = _data_indexVector + 12;
      const int32_t dir = *((int32_t *)(& _data_indexVector_112[16*blockDim.x*blockIdx.x + 16*threadIdx.x]));
      double * _data_pdfsf9cc34cc4e2b6261 = _data_pdfs + _stride_pdfs_1*y + _stride_pdfs_1*cy[dir] + _stride_pdfs_2*z + _stride_pdfs_2*cz[dir] + _stride_pdfs_3*invdir[dir];
      double * _data_pdfs_10_2011ac6bf6446d4afa = _data_pdfs + _stride_pdfs_1*y + _stride_pdfs_2*z + _stride_pdfs_3*dir;
      _data_pdfsf9cc34cc4e2b6261[_stride_pdfs_0*x + _stride_pdfs_0*cx[dir]] = _data_pdfs_10_2011ac6bf6446d4afa[_stride_pdfs_0*x];
   } 
}
}

#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

#ifdef __HIPCC__
#pragma pop
#endif


void UniformGridGPU_NoSlip::run( IBlock * block, IndexVectors::Type type , hipStream_t stream )
{
    auto * indexVectors = block->getData<IndexVectors>(indexVectorID);

    auto pointer = indexVectors->pointerGpu(type);
    

    int64_t indexVectorSize = int64_c( indexVectors->indexVector(type).size() );
    if( indexVectorSize == 0)
        return;

    uint8_t * _data_indexVector = reinterpret_cast<uint8_t*>(pointer);

    auto pdfs = block->getData< cuda::GPUField<double> >(pdfsID);

    WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()));
    double * _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
    const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
    const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
    const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
    const int64_t _stride_pdfs_3 = int64_t(pdfs->fStride());
    dim3 _block(int(((256 < indexVectorSize) ? 256 : indexVectorSize)), int(1), int(1));
    dim3 _grid(int(( (indexVectorSize) % (((256 < indexVectorSize) ? 256 : indexVectorSize)) == 0 ? (int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize)) : ( (int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize)) ) +1 )), int(1), int(1));
    internal_boundary_UniformGridGPU_NoSlip::boundary_UniformGridGPU_NoSlip<<<_grid, _block, 0, stream>>>(_data_indexVector, _data_pdfs, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, indexVectorSize);
}

void UniformGridGPU_NoSlip::operator() ( IBlock * block, hipStream_t stream  )
{
    run( block, IndexVectors::ALL, stream );
}

void UniformGridGPU_NoSlip::inner( IBlock * block, hipStream_t stream  )
{
    run( block, IndexVectors::INNER, stream  );
}

void UniformGridGPU_NoSlip::outer( IBlock * block, hipStream_t stream  )
{
    run( block, IndexVectors::OUTER, stream  );
}


} // namespace lbm
} // namespace walberla

